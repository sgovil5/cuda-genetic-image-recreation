#include "hip/hip_runtime.h"
#include "random_utils.cuh"

__global__ void init_curand_states(hiprandState* states, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}
