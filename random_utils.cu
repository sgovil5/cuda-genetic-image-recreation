#include "hip/hip_runtime.h"
#include "random_utils.cuh"

__global__ void init_curand_states(hiprandState* states, unsigned long long seed, int total_threads) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= total_threads) return;
    hiprand_init(seed, idx, 0, &states[idx]);
}
