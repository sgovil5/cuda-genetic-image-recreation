#include "main.cuh"

#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"



int main(){
    hipSetDevice(0);
    int width=WIDTH, height=HEIGHT, channels;
    
    unsigned char* image_data = stbi_load("image.png", &width, &height, &channels, 3);

    if (!image_data) {
        std::cerr << "Error in loading the image" << std::endl;
        exit(1);
    }

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing
    hipEventRecord(start);

    Image original_image = {image_data, width, height, channels};

    thrust::host_vector<InitialImage> population = init_population(original_image);

    thrust::host_vector<Image> image_buffers;
    for(int i=0; i<POPULATION_SIZE; i++){
        InitialImage curr_img = population[i];
        unsigned char* image_buffer = visualize_image(curr_img, WIDTH, HEIGHT);
        image_buffers.push_back({image_buffer, WIDTH, HEIGHT, channels});
        // stbi_write_png(("output_image_" + std::to_string(i) + ".png").c_str(), WIDTH, HEIGHT, channels, image_buffer, WIDTH * channels);
    }

    thrust::host_vector<float> fitness_scores = calculate_fitness(image_buffers, original_image);

    thrust::host_vector<Image> new_population = tournament_selection(image_buffers, fitness_scores);

    introduce_mutation(new_population);

    for(int i=0; i<EPOCHS; i++){
        fitness_scores = calculate_fitness(new_population, original_image);
        new_population = tournament_selection(new_population, fitness_scores);
        introduce_mutation(new_population);
        if(i%100==0){
            int max_fitness_index = thrust::max_element(fitness_scores.begin(), fitness_scores.end()) - fitness_scores.begin();
            std::cout<<"Epoch: "<<i<<" Fitness: "<<fitness_scores[max_fitness_index]<<std::endl;
            stbi_write_png(("output_image_" + std::to_string(i) + ".png").c_str(), WIDTH, HEIGHT, channels, new_population[max_fitness_index].data, WIDTH * channels);
        }
    }

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate and print the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Total execution time: %f ms\n", milliseconds);

    // Clean up CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}