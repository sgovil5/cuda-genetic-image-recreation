#include "hip/hip_runtime.h"
#include "main.cuh"

#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"



int main(){
    hipSetDevice(0);
    int width=WIDTH, height=HEIGHT, channels;
    
    unsigned char* image_data = stbi_load("image.png", &width, &height, &channels, 3);

    if (!image_data) {
        std::cerr << "Error in loading the image" << std::endl;
        exit(1);
    }

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing
    hipEventRecord(start);

    Image original_image = {image_data, width, height, channels};

    thrust::host_vector<InitialImage> population = init_population(original_image);

    thrust::host_vector<Image> image_buffers;
    for(int i=0; i<POPULATION_SIZE; i++){
        InitialImage curr_img = population[i];
        unsigned char* image_buffer = visualize_image(curr_img, WIDTH, HEIGHT);
        image_buffers.push_back({image_buffer, WIDTH, HEIGHT, channels});
        // stbi_write_png(("output_image_" + std::to_string(i) + ".png").c_str(), WIDTH, HEIGHT, channels, image_buffer, WIDTH * channels);
    }

    thrust::host_vector<float> fitness_scores = calculate_fitness(image_buffers, original_image);

    thrust::host_vector<Image> new_population = tournament_selection(image_buffers, fitness_scores);

    introduce_mutation(new_population);

    for(int i = 0; i < min(POPULATION_SIZE, 10); i++) {
        stbi_write_png(("new_output_image_" + std::to_string(i) + ".png").c_str(), 
                       WIDTH, HEIGHT, channels, new_population[i].data, WIDTH * channels);
    }


    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate and print the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Total execution time: %f ms\n", milliseconds);

    // Clean up CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}