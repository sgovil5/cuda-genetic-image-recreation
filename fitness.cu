#include "hip/hip_runtime.h"
#include "fitness.cuh"

/*
Color Utils taken from https://github.com/ThunderStruct/Color-Utilities/blob/master/ColorUtils.cpp
Modified to work with CUDA (I just added __device__ lmfao)
*/

__device__ ColorUtils::xyzColor rgbToXyz(ColorUtils::rgbColor c){
    float x, y, z, r, g, b;
    
    r = c.r / 255.0; g = c.g / 255.0; b = c.b / 255.0;
    
    if (r > 0.04045)
        r = powf(( (r + 0.055) / 1.055 ), 2.4);
    else r /= 12.92;
    
    if (g > 0.04045)
        g = powf(( (g + 0.055) / 1.055 ), 2.4);
    else g /= 12.92;
    
    if (b > 0.04045)
        b = powf(( (b + 0.055) / 1.055 ), 2.4);
    else b /= 12.92;
    
    r *= 100; g *= 100; b *= 100;
    
    // Calibration for observer @2° with illumination = D65
    x = r * 0.4124 + g * 0.3576 + b * 0.1805;
    y = r * 0.2126 + g * 0.7152 + b * 0.0722;
    z = r * 0.0193 + g * 0.1192 + b * 0.9505;
    
    return ColorUtils::xyzColor(x, y, z);
}

__device__ ColorUtils::CIELABColorSpace xyzToLab(ColorUtils::xyzColor c){
    float x, y, z, l, a, b;
    const float refX = 95.047, refY = 100.0, refZ = 108.883;
    
    // References set at calibration for observer @2° with illumination = D65
    x = c.x / refX; y = c.y / refY; z = c.z / refZ;
    
    if (x > 0.008856)
        x = powf(x, 1 / 3.0);
    else x = (7.787 * x) + (16.0 / 116.0);
    
    if (y > 0.008856)
        y = powf(y, 1 / 3.0);
    else y = (7.787 * y) + (16.0 / 116.0);
    
    if (z > 0.008856)
        z = powf(z, 1 / 3.0);
    else z = (7.787 * z) + (16.0 / 116.0);
    
    l = 116 * y - 16;
    a = 500 * (x - y);
    b = 200 * (y - z);
    
    return ColorUtils::CIELABColorSpace(l, a, b);
}

__device__ float getColorDeltaE(ColorUtils::rgbColor c1, ColorUtils::rgbColor c2) {
    ColorUtils::xyzColor xyzC1 = rgbToXyz(c1), xyzC2 = rgbToXyz(c2);
    ColorUtils::CIELABColorSpace labC1 = xyzToLab(xyzC1), labC2 = xyzToLab(xyzC2);
    
    float deltaE = sqrtf(powf(labC1.l - labC2.l, 2) + powf(labC1.a - labC2.a, 2) + powf(labC1.b - labC2.b, 2));
    
    return deltaE;
}

__global__ void calculate_fitness_kernel(Image* population, Image* original_image, float* fitness_scores) {
    int img_idx = blockIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.z * blockDim.x + threadIdx.x;

    if (img_idx >= POPULATION_SIZE || row >= HEIGHT || col >= WIDTH) return;

    int pixel_idx = row * WIDTH + col;

    unsigned char* pop_data = population[img_idx].data;
    unsigned char* orig_data = original_image->data;

    ColorUtils::rgbColor c1 = {
        pop_data[pixel_idx * 3],
        pop_data[pixel_idx * 3 + 1],
        pop_data[pixel_idx * 3 + 2]
    };

    ColorUtils::rgbColor c2 = {
        orig_data[pixel_idx * 3],
        orig_data[pixel_idx * 3 + 1],
        orig_data[pixel_idx * 3 + 2]
    };

    float deltaE = getColorDeltaE(c1, c2);
    atomicAdd(&fitness_scores[img_idx], deltaE);
}

thrust::host_vector<float> calculate_fitness(thrust::host_vector<Image>& population, Image& original_image) {
    Image* d_population;
    Image* d_original_image;
    float* d_fitness_scores;

    hipMalloc(&d_population, sizeof(Image) * POPULATION_SIZE);
    hipMalloc(&d_original_image, sizeof(Image));
    hipMalloc(&d_fitness_scores, sizeof(float) * POPULATION_SIZE);

    // Allocate and copy image data to create a deep copy for GPU usage
    for (int i = 0; i < POPULATION_SIZE; i++) {
        unsigned char* d_data;
        hipMalloc(&d_data, WIDTH * HEIGHT * 3 * sizeof(unsigned char));
        hipMemcpy(d_data, population[i].data, WIDTH * HEIGHT * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
        population[i].data = d_data;
    }

    unsigned char* d_orig_data;
    hipMalloc(&d_orig_data, WIDTH * HEIGHT * 3 * sizeof(unsigned char));
    hipMemcpy(d_orig_data, original_image.data, WIDTH * HEIGHT * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
    original_image.data = d_orig_data;

    hipMemcpy(d_population, population.data(), sizeof(Image) * POPULATION_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_original_image, &original_image, sizeof(Image), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);  // 2D thread block
    dim3 gridDim(
        POPULATION_SIZE,
        (original_image.height + threadsPerBlock.y - 1) / threadsPerBlock.y,
        (original_image.width + threadsPerBlock.x - 1) / threadsPerBlock.x
    );

    calculate_fitness_kernel<<<gridDim, threadsPerBlock>>>(d_population, d_original_image, d_fitness_scores);
    
    hipDeviceSynchronize();

    thrust::host_vector<float> fitness_scores(POPULATION_SIZE);
    hipMemcpy(fitness_scores.data(), d_fitness_scores, sizeof(float) * POPULATION_SIZE, hipMemcpyDeviceToHost);

    // Clean up
    for (int i = 0; i < POPULATION_SIZE; i++) {
        hipFree(population[i].data);
    }
    hipFree(d_orig_data);
    hipFree(d_population);
    hipFree(d_original_image);
    hipFree(d_fitness_scores);

    return fitness_scores;
}