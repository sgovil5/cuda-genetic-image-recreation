#include "hip/hip_runtime.h"
#include "initialize.cuh"

Color calculate_avg_color(Image image){
    long long total_r = 0, total_g = 0, total_b = 0;
    int total_pixels = image.width * image.height;

    for (int i = 0; i < total_pixels * image.channels; i += image.channels) {
        total_r += image.data[i];
        total_g += image.data[i + 1];
        total_b += image.data[i + 2];
    }

    return Color{
        static_cast<float>(total_r / total_pixels),
        static_cast<float>(total_g / total_pixels),
        static_cast<float>(total_b / total_pixels),
        255
    };
}

__device__ Polygon create_polygon(hiprandState* state) {
    Polygon polygon;
    polygon.color = {
        device_uniform_dis(state) * 255,
        device_uniform_dis(state) * 255,
        device_uniform_dis(state) * 255,
        device_uniform_dis(state) * 255
    };

    int num_points = hiprand_uniform(state) * 3 + 3; // 3-6 points
    polygon.num_points = num_points;

    for (int i = 0; i < num_points; i++) {
        polygon.points[i].x = device_uniform_dis(state) * (WIDTH - 1);
        polygon.points[i].y = device_uniform_dis(state) * (HEIGHT - 1);
    }

    for(int i=0; i<num_points; i++){
        polygon.lines[i].p1 = polygon.points[i];
        polygon.lines[i].p2 = polygon.points[(i+1)%num_points];
    }

    return polygon;
}

__global__ void generate_image_kernel(InitialImage* population, Color avg_color, hiprandState* states, unsigned char* image_buffers){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx >= POPULATION_SIZE) return;
    hiprandState local_state = states[idx];

    InitialImage img;

    float background_prob = device_uniform_dis(&local_state);
    if (background_prob < 0.4f) img.background = avg_color;
    else if (background_prob < 0.6f) img.background = {0, 0, 0, 255};
    else if (background_prob < 0.8f) img.background = {255, 255, 255, 255};
    else{
        img.background = {
            device_uniform_dis(&local_state)*255,
            device_uniform_dis(&local_state)*255,
            device_uniform_dis(&local_state)*255,
            255
        };
    }

    img.num_polygons = hiprand_uniform(&local_state) * 3 + 3; // 3-6 polygons
    
    for(int i=0; i<img.num_polygons; i++){
        img.polygons[i] = create_polygon(&local_state);
    }

    population[idx] = img;
}

thrust::host_vector<InitialImage> init_population(Image original_image){
    // Calculate average image color
    Color avg_color = calculate_avg_color(original_image);

    InitialImage* d_population;
    hiprandState* d_states;
    unsigned char* d_image_buffers;
    

    hipMalloc(&d_population, POPULATION_SIZE*sizeof(InitialImage));
    hipMalloc(&d_states, POPULATION_SIZE*sizeof(hiprandState));
    hipMalloc(&d_image_buffers, POPULATION_SIZE*WIDTH*HEIGHT*3*sizeof(unsigned char));

    CUDA_CHECK(hipMalloc(&d_population, POPULATION_SIZE*sizeof(InitialImage)));
    CUDA_CHECK(hipMalloc(&d_states, POPULATION_SIZE*sizeof(hiprandState)));
    CUDA_CHECK(hipMalloc(&d_image_buffers, POPULATION_SIZE*WIDTH*HEIGHT*3*sizeof(unsigned char)));

    int block_size = 256;
    int grid_size = (POPULATION_SIZE + block_size - 1) / block_size;

    init_curand_states<<<grid_size, block_size>>>(d_states, unsigned(time(NULL)), POPULATION_SIZE);

    generate_image_kernel<<<grid_size, block_size>>>(d_population, avg_color, d_states, d_image_buffers);

    thrust::host_vector<InitialImage> population(POPULATION_SIZE);
    hipMemcpy(population.data(), d_population, POPULATION_SIZE*sizeof(InitialImage), hipMemcpyDeviceToHost);

    hipFree(d_population);
    hipFree(d_states);
    hipFree(d_image_buffers);

    return population;
}